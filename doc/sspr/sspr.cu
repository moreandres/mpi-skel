#include "hip/hip_runtime.h"
#include <errno.h>
#include <float.h>
#include <math.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

/** SSPR - performs the symmetric rank 1 operation A := alpha*x*x**T + A */

/** print both to console and log file */

void print(const char *fmt, ...)
{
	va_list args;
	va_start(args, fmt);

	vprintf(fmt, args);
	FILE *file = fopen("sspr.log", "a");
	vfprintf(file, fmt, args);

	va_end(args);
}

static struct hipDeviceProp_t capabilities;
static size_t cudaMemGetInfo_free = 0;

/**
 * display error and exit
 *
 * \param msg error message
 * \param ret error code
 * \param desc error description
 */

static int err(char *msg, int ret, const char *desc)
{
	fprintf(stderr, msg, ret, desc);
	exit(EXIT_FAILURE);
}

static char *cublas2str[] = {
	"HIPBLAS_STATUS_SUCCESS", /* 0 */
	"HIPBLAS_STATUS_NOT_INITIALIZED",
	"unknown",
	"unknown",
	"unknown",
	"unknown", /* 5 */
	"unknown",
	"HIPBLAS_STATUS_INVALID_VALUE",
	"unknown",
	"unknown",
	"unknown", /* 10 */
	"HIPBLAS_STATUS_MAPPING_ERROR",
};

#define PACKED_SIZE(n) (n * (n + 1) / 2)

#define PRINT(field, format) printf(#field " = " format "\n", field);
#define PRINT_DIV(field, format, divisor)			\
	printf(#field " = " format "\n", field / divisor);
#define PRINT_DIM(field, format)					\
	printf(#field " = " format "\n", field[0], field[1], field[2]);

/** query CUDA device capabilities */

static void cuda_capabilities(void)
{
	hipError_t ret = hipGetDeviceProperties(&capabilities, 0);
	if (ret == hipSuccess) {
		printf("\nDEVICE\n------\n");

		PRINT(capabilities.name, "%s");
		PRINT_DIV(capabilities.totalGlobalMem,
			  "%.2f MB",
			  1024.0f / 1024.0f);
		PRINT_DIV(capabilities.sharedMemPerBlock, "%.2f KB", 1024.0f);
		PRINT(capabilities.regsPerBlock, "%d"); 
		PRINT(capabilities.warpSize, "%d");
		PRINT_DIV(capabilities.memPitch, "%.2f KB", 1024.0f);
		PRINT(capabilities.maxThreadsPerBlock, "%d");
		PRINT_DIM(capabilities.maxThreadsDim, "%d %d %d");
		PRINT_DIM(capabilities.maxGridSize, "%d %d %d");
		PRINT_DIV(capabilities.totalConstMem, "%.2f KB", 1024.0f);
		PRINT(capabilities.major, "%d");
		PRINT(capabilities.minor, "%d");
		PRINT_DIV(capabilities.clockRate, "%.2f MHz", 1024.0f);
		PRINT(capabilities.textureAlignment, "%d");
		PRINT(capabilities.deviceOverlap, "%d");
		PRINT(capabilities.multiProcessorCount, "%d");

		ret = hipMemGetInfo(&cudaMemGetInfo_free, NULL);
		if (ret != hipSuccess)
			err("hipMemGetInfo: %d (%s)",
			    ret, hipGetErrorString(ret));
		printf("hipMemGetInfo.free = %d MB\n",
		       cudaMemGetInfo_free / 1024 / 1024);
		printf("\n");
	} else
		err("hipGetDeviceProperties: %d (%s)",
		    ret, hipGetErrorString(ret));
}

/** gettimeofday() is not available in Windows, timezone not supported */

#define WINDOWS 1
#if WINDOWS == 1

#include <windows.h>

struct timezone 
{
	int tz_minuteswest; /* minutes W of Greenwich */
	int tz_dsttime;     /* type of dst correction */
};

static int gettimeofday(struct timeval *tv, struct timezone *tz)
{
	FILETIME ft;
	unsigned __int64 tmpres = 0;
 
	if (NULL != tv) {
		GetSystemTimeAsFileTime(&ft);
 
		tmpres |= ft.dwHighDateTime;
		tmpres <<= 32;
		tmpres |= ft.dwLowDateTime;

#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
#define DELTA_EPOCH_IN_MICROSECS  11644473600000000Ui64
#else
#define DELTA_EPOCH_IN_MICROSECS  11644473600000000ULL
#endif
		tmpres -= DELTA_EPOCH_IN_MICROSECS; 
#undef DELTA_EPOCH_IN_MICROSECS

		tmpres /= 10;

		tv->tv_sec = (long) (tmpres / 1000000UL);
		tv->tv_usec = (long) (tmpres % 1000000UL);
	}
  
	return 0;
}
#endif /* WINDOWS */

/** get time in seconds */

static double wtime(void)
{
	struct timeval tv;
  	gettimeofday(&tv, NULL);

	return tv.tv_sec + tv.tv_usec / 1000000.0;
}

#define UPPER 1

/**
 * CPU naive implementation of SSPR
 *
 * Based on http://www.netlib.org/blas/sspr.f and GSL's source_spr.h
 * Only UPPER mode with incx == 1 is implemented
 */

static int sspr_cpu(int uplo,
		    int n,
		    const float alpha,
		    const float *x,
		    int incx,
		    float *ap)
{
	/* parameter check */
	if ((uplo != UPPER) ||
	    n < 0 ||
	    alpha == NULL || 
	    x == NULL ||
	    incx <= 0 ||
	    ap == NULL)
		return EINVAL;

	/* return if nothing to do */
	if (n == 0 || alpha == 0.0)
		return EXIT_SUCCESS;

	/* only UPPER mode with incx == 1 is implemented */
	if (uplo != UPPER || incx != 1)
		return ENOTSUP;

	int i = 0;
	for (i = 0; i < n; i++) {
		const float tmp = alpha * x[i];

		int j = 0;
		for (j = 0; j <= i; j++)
			ap[((i*(i+1))/ 2 + j)] += x[j] * tmp;
	}

	return EXIT_SUCCESS;
}

/** CUBLAS's SSPR call */

static int sspr_cublas(int uplo,
		       int n,
		       const float alpha,
		       const float *x,
		       int incx,
		       float *ap)
{
	hipError_t ret = hipSuccess;

	hipblasHandle_t handle = NULL;
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasCreate: %d (%s)", status, cublas2str[status]);

	hipblasFillMode_t mode = (uplo == UPPER) ?
		HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
	
	int packed_n = PACKED_SIZE(n);

	float *cx = NULL;
	ret = hipMalloc(&cx, n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cx): %d (%s)", ret, hipGetErrorString(ret));
		
	status = hipblasSetVector(n, sizeof(float), x, 1, cx, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("cudaSetVector (x): %d (%s)", status, cublas2str[status]);

	float *cap = NULL;
	ret = hipMalloc(&cap, packed_n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cap): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasSetVector(packed_n, sizeof(float), ap, 1, cap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasSetVector (ap): %d (%s)", status, cublas2str[status]);

	status = hipblasSspr(handle, mode, n, &alpha, cx, incx, cap);
	if (status != HIPBLAS_STATUS_SUCCESS)
	 	err("hipblasSspr: %d (%s)", status, cublas2str[status]);

	status = hipblasGetVector(packed_n, sizeof(float), cap, 1, ap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasGetVector (ap): %d (%s)", status, cublas2str[status]);
			
	ret = hipFree(cap);
	if (ret != hipSuccess)
		err("hipFree (cap): %d (%s)", ret, hipGetErrorString(ret));

	ret = hipFree(cx);
	if (ret != hipSuccess)
		err("hipFree (cx): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
                err("hipblasDestroy: %d (%s)", status, cublas2str[status]);

	return EXIT_SUCCESS;
}

__global__ void sspr_naive_kernel(int uplo, int n, float alpha,
				  const float *x, int incx, float *ap)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		const float tmp = alpha * x[i];

		int j = 0;
		for (j = 0; j <= i; j++)
			ap[((i*(i+1))/ 2 + j)] += x[j] * tmp;
	}
}

static int sspr_naive(int uplo,
		      int n,
		      const float alpha,
		      const float *x,
		      int incx,
		      float *ap)
{
	hipError_t ret = hipSuccess;

	hipblasHandle_t handle = NULL;
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasCreate: %d (%s)", status, cublas2str[status]);
	
	int packed_n = PACKED_SIZE(n);

	float *cx = NULL;
	ret = hipMalloc(&cx, n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cx): %d (%s)", ret, hipGetErrorString(ret));
		
	status = hipblasSetVector(n, sizeof(float), x, 1, cx, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("cudaSetVector (x): %d (%s)", status, cublas2str[status]);

	float *cap = NULL;
	ret = hipMalloc(&cap, packed_n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cap): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasSetVector(packed_n, sizeof(float), ap, 1, cap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasSetVector (ap): %d (%s)", status, cublas2str[status]);

	sspr_naive_kernel<<< (n / capabilities.maxThreadsPerBlock),
		(capabilities.maxThreadsPerBlock) >>>
		(uplo, n, alpha, cx, incx, cap);

	status = hipblasGetVector(packed_n, sizeof(float), cap, 1, ap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasGetVector (ap): %d (%s)", status, cublas2str[status]);
			
	ret = hipFree(cap);
	if (ret != hipSuccess)
		err("hipFree (cap): %d (%s)", ret, hipGetErrorString(ret));

	ret = hipFree(cx);
	if (ret != hipSuccess)
		err("hipFree (cx): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
                err("hipblasDestroy: %d (%s)", status, cublas2str[status]);

	return EXIT_SUCCESS;
}

__global__ void sspr_optimized_kernel(int uplo, int n, float alpha,
				      const float *x, int incx, float *ap)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {

		int tid = threadIdx.x;
		
		extern __shared__ float cache[];
		float *xi = (float *) cache;

		const float tmp = alpha * x[i];
		xi[tid] = x[i];

		int j = 0;
		for (j = 0; j <= i; j++) {
			ap[((i*(i+1))/ 2 + j)] += x[j] * tmp;
		}
	}
}

static int sspr_optimized(int uplo,
			  int n,
			  const float alpha,
			  const float *x,
			  int incx,
			  float *ap)
{
	hipError_t ret = hipSuccess;

	hipblasHandle_t handle = NULL;
	hipblasStatus_t status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasCreate: %d (%s)", status, cublas2str[status]);
	
	int packed_n = PACKED_SIZE(n);

	float *cx = NULL;
	ret = hipMalloc(&cx, n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cx): %d (%s)", ret, hipGetErrorString(ret));
		
	status = hipblasSetVector(n, sizeof(float), x, 1, cx, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("cudaSetVector (x): %d (%s)", status, cublas2str[status]);

	float *cap = NULL;
	ret = hipMalloc(&cap, packed_n * sizeof(float));
	if (ret != hipSuccess)
		err("hipMalloc (cap): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasSetVector(packed_n, sizeof(float), ap, 1, cap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasSetVector (ap): %d (%s)", status, cublas2str[status]);

	sspr_optimized_kernel<<< (n / capabilities.maxThreadsPerBlock),
		(capabilities.maxThreadsPerBlock) >>>
		(uplo, n, alpha, cx, incx, cap);

	status = hipblasGetVector(packed_n, sizeof(float), cap, 1, ap, 1);
	if (status != HIPBLAS_STATUS_SUCCESS)
		err("hipblasGetVector (ap): %d (%s)", status, cublas2str[status]);
			
	ret = hipFree(cap);
	if (ret != hipSuccess)
		err("hipFree (cap): %d (%s)", ret, hipGetErrorString(ret));

	ret = hipFree(cx);
	if (ret != hipSuccess)
		err("hipFree (cx): %d (%s)", ret, hipGetErrorString(ret));

	status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
                err("hipblasDestroy: %d (%s)", status, cublas2str[status]);

	return EXIT_SUCCESS;
}

/** reduce all elements of a packed matrix of size n */

static float reduce(float *ap, int n)
{
	float res = 0.0;

	int i = 0;
	for (i = 0; i < n; i++)
		res += ap[i];

	return res;
}

static void frand(float *array, int size)
{
	srand(size);

	int i = 0;
	for (i = 0; i < size; i++)
		array[i] = 1.0f * rand() / RAND_MAX;
}

#define SSPR_N_DEFAULT (1 << 12)
#define SSPR_ALPHA_DEFAULT (3.14159265358979323846)

int main(int argc, char *argv[])
{
	cuda_capabilities();

	int n = ( getenv("SSPR_N") ) ? atoi(getenv("SSPR_N")) : SSPR_N_DEFAULT;

	const float alpha =
		(getenv("SSPR_ALPHA")) ?
		atof(getenv("SSPR_ALPHA")) : SSPR_ALPHA_DEFAULT;

	int packed_n = PACKED_SIZE(n);

	printf("INPUT\n-----\nSSPR_N = %d floats (packed %d floats)\nSSPR_ALPHA = %f\n",
	       n, packed_n, alpha);

	int x_bytes = n * sizeof(float);
	int ap_bytes = packed_n * sizeof(float);

	printf("memory = %d MB\n", (x_bytes + ap_bytes ) / 1024 / 1024);

	float *x = (float *) malloc(x_bytes);
	if (!x)
		err("malloc (x): %d (%s)", errno, strerror(errno));
	float *ap = (float *) malloc(ap_bytes);
	if (!ap)
		err("malloc (ap): %d (%s)", errno, strerror(errno));

	printf("\nSSPR\n----\n");

	int k = 0;
	for (k = 0; k < 1; k++) {
		{
			frand(x, n);
			frand(ap, packed_n);
			double start = wtime();			
			int ret = sspr_cpu(UPPER, n, alpha, x, 1, ap);
			double end = wtime();
			print("%s, %d, %f, %d, %f\n",
			      "sspr_cpu", n, end - start, ret, reduce(ap, packed_n));
		}
		{
			frand(x, n);
			frand(ap, packed_n);
			double start = wtime();			
			int ret = sspr_cublas(UPPER, n, alpha, x, 1, ap);
			double end = wtime();
			print("%s, %d, %f, %d, %f\n",
			      "sspr_cublas", n, end - start, ret, reduce(ap, packed_n));
		}
		{
			frand(x, n);
			frand(ap, packed_n);
			double start = wtime();			
			int ret = sspr_naive(UPPER, n, alpha, x, 1, ap);
			double end = wtime();
			print("%s, %d, %f, %d, %f\n",
			      "sspr_naive", n, end - start, ret, reduce(ap, packed_n));
		}
		{
			frand(x, n);
			frand(ap, packed_n);
			double start = wtime();			
			int ret = sspr_optimized(UPPER, n, alpha, x, 1, ap);
			double end = wtime();
			print("%s, %d, %f, %d, %f\n",
			      "sspr_optimized", n, end - start, ret, reduce(ap, packed_n));
		}
	}
	printf("\n");

	return 0;
}
